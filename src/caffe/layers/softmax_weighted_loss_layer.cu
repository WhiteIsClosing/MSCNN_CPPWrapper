#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxWeightedLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, const Dtype* weight_map, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN))) * weight_map[n * spatial_dim + s];
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithWeightedLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;

  // setup weighting map
  const int bottom_num = bottom.size();
  Dtype* weight_map_gpu = weight_map_.mutable_gpu_data();
  caffe_gpu_set(weight_map_.count(), Dtype(1), weight_map_gpu);
  if (bottom_num == 3) {
    CHECK_EQ(outer_num_*inner_num_,bottom[2]->count());
    CHECK_EQ(weight_map_.count(),bottom[2]->count());
    const Dtype* sample_weights_data = bottom[2]->gpu_data();
    caffe_gpu_memcpy(bottom[2]->count() * sizeof(Dtype), sample_weights_data, 
                     weight_map_gpu);
  }

  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  const Dtype* weight_map = weight_map_.gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SoftmaxWeightedLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, weight_map, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  if (normalize_) {
    Dtype count;
    caffe_gpu_asum(nthreads, counts, &count);
    loss /= count;
  } else {
    loss /= outer_num_;
  }
  top[0]->mutable_cpu_data()[0] = loss;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void SoftmaxWeightedLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithWeightedLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const Dtype* weight_map = weight_map_.gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxWeightedLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);

    // weihgting the class gradient
    for (int i = 0; i < outer_num_; i++) {
      for (int j = 0; j < bottom[0]->shape(softmax_axis_); j++) {
        Dtype* tmp_diff = bottom_diff+i*dim+j*inner_num_;
        caffe_gpu_mul<Dtype>(inner_num_, tmp_diff, weight_map+i*inner_num_, tmp_diff);
      }
    }

    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      Dtype count;
      caffe_gpu_asum(nthreads, counts, &count);
      caffe_gpu_scal(prob_.count(), loss_weight / count, bottom_diff);
    } else {
      caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithWeightedLossLayer);

}  // namespace caffe
